#include "hip/hip_runtime.h"
// Copyright 2016-present, Facebook, Inc.
// All rights reserved.
//
// This source code is licensed under the license found in the
// LICENSE file in the root directory of this source tree.

#include "Convolution.h"
#include "RuleBookIterator.h"

template <typename T, Int Dimension>
double cuda_Convolution_updateOutput(
    /*long*/ at::Tensor inputSize, /*long*/ at::Tensor outputSize,
    /*long*/ at::Tensor filterSize,
    /*long*/ at::Tensor filterStride, Metadata<Dimension> &m,
    /*cuda float*/ at::Tensor input_features,
    /*cuda float*/ at::Tensor output_features, /*cuda float*/ at::Tensor weight,
    /*cuda float*/ at::Tensor bias) {

  auto _rules =
      m.getRuleBook(inputSize, outputSize, filterSize, filterStride, true);
  Int nActive = m.getNActive(outputSize);
  output_features.resize_({nActive, weight.size(2)});
  if (not bias.numel())
    output_features.zero_();

  double flops = 0;
  if (nActive) {
    auto iF = input_features.data<T>();
    auto oF = output_features.data<T>();
    Int ip = input_features.size(1);
    Int op = output_features.size(1);
    auto w = weight.data<T>();

    if (bias.numel()) {
      auto b = bias.data<T>();
      for (Int i = 0; i < op; i += 32) {
        Int blockDim = min((Int)32, op - i);
        Int gridDim = min((Int)4096, nActive);
        Convolution_fp_bias<<<gridDim, blockDim>>>(oF + i, b + i, op, op,
                                                   nActive);
      }
    }
    Int c = ip * op;
    RULEBOOKITERATOR(
        dConvolution_forward2<T>(iF, oF, w, rbB, nHotB, ip, ip, op, op);
        , w += c; flops += nHotB * c;)
  }
  return flops;
}

template <typename T, Int Dimension>
void cuda_Convolution_backward(
    /*long*/ at::Tensor inputSize, /*long*/ at::Tensor outputSize,
    /*long*/ at::Tensor filterSize,
    /*long*/ at::Tensor filterStride, Metadata<Dimension> &m,
    /*cuda float*/ at::Tensor input_features,
    /*cuda float*/ at::Tensor d_input_features,
    /*cuda float*/ at::Tensor d_output_features,
    /*cuda float*/ at::Tensor weight, /*cuda float*/ at::Tensor d_weight,
    /*cuda float*/ at::Tensor d_bias) {

  auto _rules =
      m.getRuleBook(inputSize, outputSize, filterSize, filterStride, true);
  Int nActive = m.getNActive(outputSize);
  d_input_features.resize_as_(input_features);
  d_input_features.zero_();

  if (nActive) {
    auto iF = input_features.data<T>();
    auto diF = d_input_features.data<T>();
    auto doF = d_output_features.data<T>();
    Int ip = input_features.size(1);
    Int op = d_output_features.size(1);
    auto w = weight.data<T>();
    auto dw = d_weight.data<T>();
    Int c = ip * op;
    RULEBOOKITERATOR(dConvolution_backward_dW2<T>(iF, diF, doF, w, dw, rbB,
                                                  nHotB, ip, ip, op, op);
                     , w += c; dw += c;)

    if (d_bias.numel()) {
      auto db = d_bias.data<T>();
      Convolution_bp_bias(doF, db, op, op, nActive);
    }
  }
}

template <typename T, Int Dimension>
double cuda_SubmanifoldConvolution_updateOutput(
    /*long*/ at::Tensor inputSize, /*long*/ at::Tensor filterSize,
    Metadata<Dimension> &m,
    /*cuda float*/ at::Tensor input_features,
    /*cuda float*/ at::Tensor output_features, /*cuda float*/ at::Tensor weight,
    /*cuda float*/ at::Tensor bias) {

  auto _rules = m.getSubmanifoldRuleBook(inputSize, filterSize, true);
  Int nActive = m.getNActive(inputSize);
  output_features.resize_({nActive, weight.size(2)});
  if (bias.numel() and nActive)
    output_features.copy_(bias);
  else
    output_features.zero_();

  double flops = 0;
  if (nActive) {
    auto iF = input_features.data<T>();
    auto oF = output_features.data<T>();
    Int ip = input_features.size(1);
    Int op = output_features.size(1);
    auto w = weight.data<T>();

    // if (bias.numel()) {
    //   auto b = bias.data<T>();
    //   for (Int i = 0; i < op; i += 32) {
    //     Int blockDim = min((Int)32, op - i);
    //     Int gridDim = min((Int)4096, nActive);
    //     Convolution_fp_bias<<<gridDim, blockDim>>>(oF + i, b + i, op, op,
    //                                                nActive);
    //   }
    // }
    Int c = ip * op;
    RULEBOOKITERATOR(
        dConvolution_forward2<T>(iF, oF, w, rbB, nHotB, ip, ip, op, op);
        , w += c; flops += nHotB * c;)
  }
  return flops;
}

template <typename T, Int Dimension>
void cuda_SubmanifoldConvolution_backward(
    /*long*/ at::Tensor inputSize, /*long*/ at::Tensor filterSize,
    Metadata<Dimension> &m,
    /*cuda float*/ at::Tensor input_features,
    /*cuda float*/ at::Tensor d_input_features,
    /*cuda float*/ at::Tensor d_output_features,
    /*cuda float*/ at::Tensor weight, /*cuda float*/ at::Tensor d_weight,
    /*cuda float*/ at::Tensor d_bias) {

  auto _rules = m.getSubmanifoldRuleBook(inputSize, filterSize, true);
  Int nActive = m.getNActive(inputSize);
  d_input_features.resize_as_(input_features);
  d_input_features.zero_();

  if (nActive) {
    auto iF = input_features.data<T>();
    auto diF = d_input_features.data<T>();
    auto doF = d_output_features.data<T>();
    Int ip = input_features.size(1);
    Int op = d_output_features.size(1);
    auto w = weight.data<T>();
    auto dw = d_weight.data<T>();
    Int c = ip * op;
    RULEBOOKITERATOR(dConvolution_backward_dW2<T>(iF, diF, doF, w, dw, rbB,
                                                  nHotB, ip, ip, op, op);
                     , w += c; dw += c;)

    if (d_bias.numel()) {
      auto db = d_bias.data<T>();
      Convolution_bp_bias(doF, db, op, op, nActive);
    }
  }
}

template <typename T, Int Dimension>
double cuda_FullConvolution_updateOutput(
    /*long*/ at::Tensor inputSize, /*long*/ at::Tensor outputSize,
    /*long*/ at::Tensor filterSize,
    /*long*/ at::Tensor filterStride, Metadata<Dimension> &mIn,
    Metadata<Dimension> &mOut,
    /*cuda float*/ at::Tensor input_features,
    /*cuda float*/ at::Tensor output_features, /*cuda float*/ at::Tensor weight,
    /*cuda float*/ at::Tensor bias) {

  auto _rules = mIn.getFullConvolutionRuleBook(inputSize, outputSize,
                                               filterSize, filterStride, mOut);
  Int nActive = mOut.getNActive(outputSize);
  output_features.resize_({nActive, weight.size(2)});
  if (not bias.numel())
    output_features.zero_();
  double flops = 0;

  if (nActive) {
    auto iF = input_features.data<T>();
    auto oF = output_features.data<T>();
    Int ip = input_features.size(1);
    Int op = output_features.size(1);
    auto w = weight.data<T>();

    if (bias.numel()) {
      auto b = bias.data<T>();
      for (Int i = 0; i < op; i += 32) {
        Int blockDim = min((Int)32, op - i);
        Int gridDim = min((Int)4096, nActive);
        Convolution_fp_bias<<<gridDim, blockDim>>>(oF + i, b + i, op, op,
                                                   nActive);
      }
    }
    Int c = ip * op;
    RULEBOOKITERATOR(
        dConvolution_forward2<T>(iF, oF, w, rbB, nHotB, ip, ip, op, op);
        , w += c; flops += nHotB * c;)
  }
  return flops;
}

template <typename T, Int Dimension>
void cuda_FullConvolution_backward(
    /*long*/ at::Tensor inputSize, /*long*/ at::Tensor outputSize,
    /*long*/ at::Tensor filterSize,
    /*long*/ at::Tensor filterStride, Metadata<Dimension> &mIn,
    Metadata<Dimension> &mOut,
    /*cuda float*/ at::Tensor input_features,
    /*cuda float*/ at::Tensor d_input_features,
    /*cuda float*/ at::Tensor d_output_features,
    /*cuda float*/ at::Tensor weight, /*cuda float*/ at::Tensor d_weight,
    /*cuda float*/ at::Tensor d_bias) {

  auto _rules = mIn.getFullConvolutionRuleBook(inputSize, outputSize,
                                               filterSize, filterStride, mOut);
  Int nActive = mOut.getNActive(outputSize);
  d_input_features.resize_as_(input_features);
  d_input_features.zero_();
  if (nActive) {
    auto iF = input_features.data<T>();
    auto diF = d_input_features.data<T>();
    auto doF = d_output_features.data<T>();
    Int ip = input_features.size(1);
    Int op = d_output_features.size(1);
    auto w = weight.data<T>();
    auto dw = d_weight.data<T>();
    Int c = ip * op;
    RULEBOOKITERATOR(dConvolution_backward_dW2<T>(iF, diF, doF, w, dw, rbB,
                                                  nHotB, ip, ip, op, op);
                     , w += c; dw += c;)

    if (d_bias.numel()) {
      auto db = d_bias.data<T>();
      Convolution_bp_bias(doF, db, op, op, nActive);
    }
  }
}
template <typename T, Int Dimension>
double cuda_RandomizedStrideConvolution_updateOutput(
    /*long*/ at::Tensor inputSize, /*long*/ at::Tensor outputSize,
    /*long*/ at::Tensor filterSize,
    /*long*/ at::Tensor filterStride, Metadata<Dimension> &m,
    /*cuda float*/ at::Tensor input_features,
    /*cuda float*/ at::Tensor output_features,
    /*cuda float*/ at::Tensor weight, /*cuda float*/ at::Tensor bias) {

  auto _rules = m.getRandomizedStrideRuleBook(inputSize, outputSize, filterSize,
                                              filterStride, true);
  Int nActive = m.getNActive(outputSize);
  output_features.resize_({nActive, weight.size(2)});
  if (not bias.numel())
    output_features.zero_();

  double flops = 0;
  if (nActive) {
    auto iF = input_features.data<T>();
    auto oF = output_features.data<T>();
    Int ip = input_features.size(1);
    Int op = output_features.size(1);
    auto w = weight.data<T>();

    if (bias.numel()) {
      auto b = bias.data<T>();
      for (Int i = 0; i < op; i += 32) {
        Int blockDim = min((Int)32, op - i);
        Int gridDim = min((Int)4096, nActive);
        Convolution_fp_bias<<<gridDim, blockDim>>>(oF + i, b + i, op, op,
                                                   nActive);
      }
    }
    Int c = ip * op;
    RULEBOOKITERATOR(
        dConvolution_forward2<T>(iF, oF, w, rbB, nHotB, ip, ip, op, op);
        , w += c; flops += nHotB * c;)
  }
  return flops;
}

template <typename T, Int Dimension>
void cuda_RandomizedStrideConvolution_backward(
    /*long*/ at::Tensor inputSize, /*long*/ at::Tensor outputSize,
    /*long*/ at::Tensor filterSize,
    /*long*/ at::Tensor filterStride, Metadata<Dimension> &m,
    /*cuda float*/ at::Tensor input_features,
    /*cuda float*/ at::Tensor d_input_features,
    /*cuda float*/ at::Tensor d_output_features,
    /*cuda float*/ at::Tensor weight, /*cuda float*/ at::Tensor d_weight,
    /*cuda float*/ at::Tensor d_bias) {

  auto _rules = m.getRandomizedStrideRuleBook(inputSize, outputSize, filterSize,
                                              filterStride, true);
  Int nActive = m.getNActive(outputSize);
  d_input_features.resize_as_(input_features);
  d_input_features.zero_();

  if (nActive) {
    auto iF = input_features.data<T>();
    auto diF = d_input_features.data<T>();
    auto doF = d_output_features.data<T>();
    Int ip = input_features.size(1);
    Int op = d_output_features.size(1);
    auto w = weight.data<T>();
    auto dw = d_weight.data<T>();
    Int c = ip * op;
    RULEBOOKITERATOR(dConvolution_backward_dW2<T>(iF, diF, doF, w, dw, rbB,
                                                  nHotB, ip, ip, op, op);
                     , w += c; dw += c;)

    if (d_bias.numel()) {
      auto db = d_bias.data<T>();
      Convolution_bp_bias(doF, db, op, op, nActive);
    }
  }
}
